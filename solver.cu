#include "hip/hip_runtime.h"
/*
	

*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "board.h"

/*	Parallel version of moveAllWalls. 
	2D thread array. 
	Input: walls, moves, opponentIdx

*/
__global__ void
CUDA_solveForAllWalls(wall *d_walls, nextMove *d_moves, int oppPos) {
	int tidx = threadIdx.x;		// X-Dim = Wall
	int tidy = threadIdx.y;		// Y-Dim = Direction
	int idx = blockIdx.x;		// Space #

	//printf("Thread: (%d, %d)\n", tidx, tidy);
	// - - - - - 
	// Coalesced Load d_walls Global --> Shared for this Block
	// Only threads (0-15, 0)
	__shared__ wall sharedWalls[NUM_WALLS];

	if (tidy == 0) {
		sharedWalls[tidx] = d_walls[tidx];
	}


	// - - - - -
	// Create a blank board template --> Shared for this block
	// Only threads (0-15, 1)
	__shared__ space sharedBoardTemplate[NUM_SPACES];

	// Spaces 0-15	First 16 spaces
	if (tidy == 1) {
		CUDA_boardInitParallel(sharedBoardTemplate, tidx);
	}
	// Spaces 16-29
	if (tidy == 2 && (tidx + 16) < NUM_SPACES) {
		//printf("tidx: %d; NUM_SPACES: %d\n", (tidx + 16), NUM_SPACES);
		CUDA_boardInitParallel(sharedBoardTemplate, (tidx + 16));
	}


	// - - - - - 
	// Create shared move, global --> shared
	// Threads (0-4, 3)
	__shared__ nextMove move;

	if (tidy == 3) {

		switch(tidx) {		// Thread [] 
			case 0:		
				move.space = d_moves[idx].space;
				break;

			case 1:
				move.playerScore = d_moves[idx].playerScore;
				break;

			case 2:
				move.oppScore = d_moves[idx].oppScore;
				break;

			case 3:
				move.wallIdx = d_moves[idx].wallIdx;
				break;

			case 4: 
				move.newDir = d_moves[idx].newDir;
				break;
		}

	}

	__syncthreads();




	// Each thread makes local copy of walls
	wall l_walls[NUM_WALLS];
	for (int i = 0; i < NUM_WALLS; i++) {
		l_walls[i] = sharedWalls[i];
	}


	// Check for wall collisions && if it's the same direction
	wall oldDir = l_walls[tidx];

	bool sameDir = (oldDir == (wall)tidy);

	l_walls[tidx] = (wall) tidy;
	bool collision = CUDA_checkWallCollisions(&l_walls[0], tidx);

	if (sameDir || collision) {
		// Do nothing and return
		return;
	}
	// If no collision, contune


	// Create local copy of new board 
	space l_board[NUM_SPACES];
	for (int i = 0; i < NUM_SPACES; i++) {
		l_board[i] = sharedBoardTemplate[i];
	}


	// Generate the board from the walls
	CUDA_generateBoard(&l_board[0], l_walls);


	// Calculate shortest path for player & opponent
	int playerScore = CUDA_shortestPath(&l_board[0], move.space);
	int oppScore = CUDA_shortestPath(&l_board[0], oppPos);

	printf("PlayerPos: %d -- Wall Idx: %d -- Wall Pos: %d -- PlayerScore: %d -- OppScore: %d\n", move.space, tidx, tidy, playerScore, oppScore);

	if (playerScore < move.playerScore || oppScore > move.oppScore) {
		move.playerScore = playerScore;
		move.oppScore = oppScore;
		move.wallIdx = tidx;
		move.newDir = (wall) tidy;
	}

	d_moves[idx] = move;

} 


// CUDA Error Check
void checkCudaError(hipError_t e, char const *in) {
	if (e != hipSuccess) {
		printf("CUDA Error: %s, %s \n", in, hipGetErrorString(e));
		exit(EXIT_FAILURE);
	}
}


int main(int argc, char const *argv[])
{
	int playerPos = 0;
	int oppPos = 0;
	
	int numSpaces = SPACE_LENGTH * SPACE_WIDTH;
	size_t spaceSize = sizeof(space) * numSpaces;

	int numWalls = WALL_LENGTH * WALL_WIDTH;
	size_t wallSize = sizeof(wall) * numWalls;


	// Malloc the array of wall / board
	wall *walls = (wall *)malloc(wallSize);
	space *board = (space *)malloc(spaceSize);

	// Initialize and setup the current board state
	boardInit(board);
	generateWalls(walls);
	generateBoard(board, walls);

	//display board
	outputBoard(board);  //display by numbers
	displayBoard(board); //display visually
	
	// Find nearest neighbors to player
	int *neighbors = findNeighbors(board, playerPos);

	// Determine the number of spaces around the player
	// Count the number of possible spaces = # of blocks
	int possibleSpaces = 0;
	for (int i = 0; i < 12; i++) {
		if (neighbors[i] != -1) {
			possibleSpaces++;
		}
	}

	// Malloc an array nextMove[ # of neighbors ]
	nextMove *moves = (nextMove *)malloc( sizeof(nextMove) * possibleSpaces );

	// Zero-out the results array and set each move.space ot the neighbor space
	int j = 0;
	for (int i = 0; i < 12 && j < possibleSpaces; i++) {
		if (neighbors[i] != -1) {
			printf("Init results array. Moves[%d], Space: %d\n", j, neighbors[i]);

			moves[j].space = neighbors[i];
			moves[j].playerScore = 100;		// Intentionally high preset
			moves[j].oppScore = -1;
			moves[j].wallIdx = -1;
			moves[j].newDir = (wall) 0;

			j++;
		}
	}




	// Malloc space on device, copy to device
	wall *d_walls = NULL;
	nextMove *d_moves = NULL;

	checkCudaError( hipMalloc((void**) &d_walls, wallSize), 
		"Malloc d_walls");
	checkCudaError( hipMalloc((void**) &d_moves, (sizeof(nextMove) * possibleSpaces) ), 
		"Malloc d_walls");

	// hipMemcpy(target, source, size, function)
	checkCudaError( hipMemcpy(d_walls, walls, wallSize, hipMemcpyHostToDevice), 
		"Copy walls to device");
	checkCudaError( hipMemcpy(d_moves, moves, (sizeof(nextMove) * possibleSpaces), hipMemcpyHostToDevice), 
		"Copy moves to device");




	// Setup: Measure Runtime
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	/*	Kernel Call
		Blocks = possible spaces
		Threads = #walls * #possible directions

	*/
	dim3 grid(16,4);
	CUDA_solveForAllWalls <<<possibleSpaces, grid>>> (d_walls, d_moves, oppPos);

	checkCudaError(hipGetLastError(), "Checking Last Error, Kernel Launch");

	// Report kernel runtime
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to generate: %0.5f seconds\n", elapsedTime/1000);

	hipEventDestroy(start);
	hipEventDestroy(stop);


	// Copy Device --> Host
	// hipMemcpy(target, source, size, function)
	checkCudaError( hipMemcpy(moves, d_moves, (sizeof(nextMove) * possibleSpaces), hipMemcpyDeviceToHost), 
		"Copy moves to host");


	outputResults(moves, possibleSpaces);

	// PICK THE BEST MOVE
	nextMove bestMove = pickBestMove(moves, possibleSpaces);
	printf("Best Move: %d\n", bestMove.space);

	// Free Memory
	checkCudaError(hipFree(d_walls), "Free device histogram");
	checkCudaError(hipFree(d_moves), "Free device atom_list");

	free(board);
	free(walls);
	free(moves);

	return 0;
}
